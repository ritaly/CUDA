#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b) // funkcja uruchamiana na karcie graficnzej  
{
	int i = threadIdx.x; // watek korzysta ze stlaej automatycznej ID - identyifkator 
	int j = blockIdx.x; // podpunkt A = 1
	c[i] = 1000 *j + i;
	/* int i = 1000 * blockIdx.x + threadIdx.x; // identyfikator w�tku (w ramach bloku)
	c[threadIdx.x] = i; */

}
// nie ma podzialu na rozne bloki! wiec bedzie tylko1 blok 
int main()
{
	const int arraySize = 3;
	const int a[arraySize] = { 0 }; // w pamieci operacyjnej komputera 
	const int b[arraySize] = { 0 };
	int c[arraySize] = { 0 }; // wypelnione smaymi 0 

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize); // dodawanei - funckja! 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("c = { lubie w dupe ");

	for (int i = 0; i < 32; i++) {
		printf("%d ", c[i]); // sprawdzimt cyz nadal s zera 
	}
	printf("}\n");
	

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset(); // zakonczenie pracy 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
// koniec kodu dla procesora 
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);  //nawiazanie komunikacji z karta o nr 0 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// wszytskie te wywolania sa synchroniczne. 
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int)); // alkoujemy pamiec wiec wskanzik przyjmuje sensowene wartosci i pod tym adresem mozemy zapsiac sensowne wartosci 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// koniec allkoowania pamieci na karcie graficnej - wczesniej bylo na pamieci operacyjnej a teraz to robimy na karcie graficznej. 
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// c nie kopiuemmy bo nie interesuja nas co bylo! 
	// Launch a kernel on the GPU with one thread for each element.
	// wszystko to powyzej sie juz wykonalo 
	int numBlocks = 4;
	addKernel << <numBlocks, size >> >(dev_c, dev_a, dev_b); // jaka struktura blokow - 1: blok jest 1, size u nas = 5 czyli tworzymy blok 5 watkowy i pozostale 27 bedzie nic nei robilo 
	// zaczynamy tutaj przetwarzac na karcie: 
	// Check for any errors launching the kernel

	//Sprawdzamy bledy uruchomienia: i mozemy sie dowiedziec czy nei dalismy zlego wielkosci bloku 
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize(); // czekanie na wszystkie wywolania ktore zostaly wyslane 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost); // pobieramy wyniki i jka je mamy to mozemy je wyswietlic! 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a); // oddawanie pamieci 
	hipFree(dev_b);

	return cudaStatus;
}
